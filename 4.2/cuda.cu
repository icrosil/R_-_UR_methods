#include "hip/hip_runtime.h"
//compile g++ -o filename file.cpp and run filename
#include <iostream>
#include <math.h>
#include "vector"
#include "../lib/alglib/src/ap.h"
#include "../lib/alglib/src/alglibmisc.h"
#include "../lib/alglib/src/alglibinternal.h"
#include "../lib/alglib/src/linalg.h"
#include "../lib/alglib/src/statistics.h"
#include "../lib/alglib/src/dataanalysis.h"
#include "../lib/alglib/src/specialfunctions.h"
#include "../lib/alglib/src/solvers.h"
#include "../lib/alglib/src/optimization.h"
#include "../lib/alglib/src/diffequations.h"
#include "../lib/alglib/src/fasttransforms.h"
#include "../lib/alglib/src/integration.h"
#include "../lib/alglib/src/interpolation.h"
#include "../lib/alglib/src/ap.cpp"
#include "../lib/alglib/src/linalg.cpp"
#include "../lib/alglib/src/alglibmisc.cpp"
#include "../lib/alglib/src/alglibinternal.cpp"
#include "../lib/alglib/src/statistics.cpp"
#include "../lib/alglib/src/dataanalysis.cpp"
#include "../lib/alglib/src/specialfunctions.cpp"
#include "../lib/alglib/src/solvers.cpp"
#include "../lib/alglib/src/optimization.cpp"
#include "../lib/alglib/src/diffequations.cpp"
#include "../lib/alglib/src/fasttransforms.cpp"
#include "../lib/alglib/src/integration.cpp"
#include "../lib/alglib/src/interpolation.cpp"
#include <string>
#include <ctime>
#include <mkl.h>
#include <stdio.h>

using namespace std;
using namespace alglib;
using namespace alglib_impl;

void outVector (vector<double> B){
    // int additions = B.size() / 4;
    int additions = 1;
    cout<< B[0] << " ";
    for (int i = additions; i < B.size() - 1; i += additions){
        cout<< B[i] << " ";
    }
    cout<< B[B.size() - 1] << " ";
    cout<<endl;
}
void outVector (double* B, int N) {
    // cout<<"in outVector"<<endl;
    // int additions = B.size() / 4;
    int additions = 1;
    cout<< B[0] << " ";
    for (int i = additions; i < N - 1; i += additions){
        cout<< B[i] << " ";
    }
    cout<< B[N - 1] << " ";
    cout<<endl;
    cout<<"out outVector"<<endl;
}
void outMatr (vector<vector<double> > A){
    // int additions = A.size() / 4;
    int additions = 1;
    outVector(A[0]);
    for (int i = additions; i < A.size() - 1; i += additions){
        outVector(A[i]);
    }
    outVector(A[A.size() - 1]);
}
double F (double x, double y, double N) {
    // cout<<x<<" "<<y<<endl;
    return (2 * sin(y) - x * x * sin(y)) / N;
}
double U (double x, double y) {
    // cout<<x<<" "<<y<<endl;
    return x * x * sin(y) + 1;
}
void readMatr (vector<vector<double> > &A){
    int sizer = (int) sqrt(A.size());
    A[0][0] = -4;
    A[A.size() - 1][A.size() - 1] = -4;
    A[0][1] = 1;
    A[A.size() - 1][A.size() - 2] = 1;
    for (int i = 1; i < A.size() - 1; i++) {
        A[i][i] = -4;
        if (!((i % sizer) == 0)) {
            A[i][i - 1] = 1;
        }
        if (!(((i - 1) % sizer) == 0)) {
            A[i][i + 1] = 1;
        }
    }
    for (int i = 0; i < A.size() - sizer; i++) {
        A[i][i + sizer] = 1;
        A[i + sizer][i] = 1;
    }
}
void readVector (vector<vector<double> >& B){
    for (int i = 0; i < B.size(); i++) {
        B[i][0] = U(i / (double) (B.size() - 1), 0);
        B[0][i] = U(0, i / (double) (B.size() - 1));
        B[B.size() - 1][i] = U(1, i / (double) (B.size() - 1));
        B[i][B.size() - 1] = U(i / (double) (B.size() - 1), 1);
    }
    for (int i = 1; i < B.size() - 1; ++i) {
        for (int j = 1; j < B.size() - 1; ++j) {
            B[i][j] = F(i / (double) (B.size() - 1), j / (double) (B.size() - 1), (B.size() - 1) * (B.size() - 1));
        }
    }
}
double* arrToRealArr (vector<vector<double> >const &A){
    double * local;
    local = new double[A.size() * A.size()];
    for (int i = 0; i < A.size(); ++i){
        for (int j = 0; j < A[i].size();++j){
            local[i * A.size() + j] = A[i][j];
        }
    }
    return local;
}
void outReal1Array (alglib::real_1d_array wr) {
    for (int i = 0; i < wr.length(); ++i )
    cout<<wr[i]<<" ";
    cout<<endl;
}
double findMaxRealArr (alglib::real_1d_array const wr) {
    double max = fabs(wr[0]);
    for (int i = 1; i < wr.length(); ++i) {
        if (fabs(wr[i]) > max) max = fabs(wr[i]);
    }
    return max;
}
double findMinRealArr (alglib::real_1d_array const wr) {
    double min = fabs(wr[0]);
    for (int i = 1; i < wr.length(); ++i) {
        if (fabs(wr[i]) < min) min = fabs(wr[i]);
    }
    return min;
}

double nextTau (vector<double> Tau, double ro0, int n, vector<double> optTau) {
    double tk = cos (( optTau[Tau.size() - 1]) * M_PI / (2 * n));
    // return Tau[0] / (1 + ro0 * tk);
    // if ((Tau[0] / (1 + ro0 * tk) - Tau[Tau.size() - 1]) > 1.) {
        // counter--;
        // return Tau[counter];
    // } else {
        // counter = Tau.size();
        return Tau[0] / (1 + ro0 * tk);
    // }
    // return 2 / (max + min + (max - min) * tk);
}
int findMaxIter (double eps, double ksi) {
    // return ceil(log (2. / eps) / (2. * sqrt(ksi)));
    return ceil(log (2. / eps) / (2. * sqrt(ksi)));
}
void firstApprSet(vector<vector<double> >& B) {
    for (int i = 0; i < B.size(); i++) {
        B[i][0] = U(i / (double) (B.size() - 1), 0);
        B[0][i] = U(0, i / (double) (B.size() - 1));
        B[B.size() - 1][i] = U(1, i / (double) (B.size() - 1));
        B[i][B.size() - 1] = U(i / (double) (B.size() - 1), 1);
    }
    for (int i = 1; i < B.size() - 1; ++i) {
        for (int j = 1; j < B.size() - 1; ++j) {
            B[i][j] = F(i / (double) (B.size() - 1), j / (double) (B.size() - 1), (B.size() - 1) * (B.size() - 1)) / 2.;
        }
    }
}
double aMulX(vector<vector<double> > A, vector<double> X, int j){
    double res = 0;
    for (int i = 0; i < A.size(); ++i){
        res += A[j][i] * X[i];
    }
    return res;
}
void decToDuo(vector<double> &duo, int maxIter) {
    // while (maxIter > 0) {
    //     duo.push_back(maxIter % 2);
    //     maxIter /= 2;
    // }
    vector<double> temp(0);
    while (maxIter > 1) {
        if (maxIter % 2 != 0) {
                temp.push_back(1);
                maxIter--;
        } else {
            temp.push_back(0);
            maxIter /= 2;
        }
    }
    for (int j = 0; j < temp.size(); j++) {
        /* code */
        duo.push_back(temp[temp.size() - j - 1]);
    }
}
void calculateOptTau(vector<double> &optTau, vector<double> duo) {
    vector<double> temp(0);
    double coord = 0;
    double number = 1;
    // duo[0] = 0;
    // duo.erase(duo.begin() + duo.size() - 1);
    for (int i = 0; i < duo.size(); i++) {
        temp.clear();
        coord = 0;
        if (duo[i] == 0) {
            number *= 2;
            if ((i + 1 == duo.size()) || (duo[i + 1] == 0)) {
                /*normal*/
                coord = 0;
            } else {
                /*4m+2*/
                coord = 2;
            }
            for (int j = 0; j < optTau.size(); j++) {
                temp.push_back(optTau[j]);
                temp.push_back(4 * optTau.size() + coord - optTau[j]);
            }
        } else {
            number += 1;
            /*+1*/
            for (int j = 0; j < optTau.size(); j++) {
                temp.push_back(optTau[j]);
            }
            temp.push_back(number);
        }
        optTau.clear();
        for (int j = 0; j < temp.size(); j++) {
            optTau.push_back(temp[j]);
        }
        outVector(optTau);
    }
}
double* aMulXVector(vector<vector<double> > A, vector<double> X){
    double *res = new double [X.size()];
    for (int j = 0; j < X.size(); j++) {
        res[j] = 0;
    }
    for (int j = 0; j < A.size(); ++j){
        for (int i = 0; i < A.size(); ++i){
            res[j] += A[j][i] * X[i];
        }
    }
    return res;
}
void Shablon(vector<vector<double> > X, double * &res) {
    // cout<<"size is "<<size;
    // double *res = new double [size];
    // cout<<"in Shabo"<<endl;
    // for (int j = 0; j < size; j++) {
    //     res[j] = 0;
    // }
    cout<<"in Shabo"<<endl;
    for (int j = 1; j < X.size() - 1; ++j) {
        for (int i = 1; i < X.size() - 1; ++i) {
            res[(i - 1) * (X.size() - 2) + (j - 1) ] = X[i + 1][j] + X[i - 1][j] + X[i][j + 1] + X[i][j - 1] - 4. * X[i][j];
        }
    }
    cout<<"out Shabo"<<endl;
    // return res;
}

/**
 * CUDA functions
 */

 #ifndef N
 #define N 50

 #endif

__device__ int barrier = N - 2;
__device__ int blocks = N - 2;
__global__ void myshab (double *temp, int n, double *all) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = index + N + 1 + 2 * (int) (index / (N - 2));
        if (index < n) {
            temp[index] = -4 * all[lindex] + all[lindex - N] + all[lindex + N] + all[lindex - 1] + all[lindex + 1];
        }
}
__global__ void mykernel (double *a, double *b, double *c, double *d, int n, int i, double *all) {
//B, Shablon, Tau, firstAppr, iteration number
//tempAppr[j] = (B[j] - aMulX(A, firstAppr, j)) * Tau[i - 1] + firstAppr[j];
    //TODO syncthreads will work, so pass needed elements, let them for and sync on every iteration
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = index + N + 1 + 2 * (int) (index / (N - 2));
    if (index < n) {
        d[index] = (-a[index] + b[index]) * c[i] + d[index];
        all[lindex] = d[index];
    }
    // if i ever need an block sync (time expensive), or create own block sync with a vector of 00,
    // fill them with 1 when block is done and go next if all 1
    /* Do whatever it is that this block does. */


   //  /* Make sure all threads in this block are actually here. */
   //  __syncthreads();
   //  /* Once we're done, decrease the value of the barrier. */
   // if ( threadIdx.x == 0 )
   //     atomicSub( &barrier , 1 );
   // 
   // /* Now wait for the barrier to be zero. */
   // if ( threadIdx.x == 0 )
   //     while ( atomicCAS( &barrier , 0 , 0 ) != 0 );
   // 
   // /* Make sure everybody has waited for the barrier. */
   // __syncthreads();
   // 
   // /* Carry on with whatever else you wanted to do. */
   // barrier = N - 2;
}



int main() {
    /**
     * t0 is for documenting whole processing time
     * @type double
     */
    double t0 = dsecnd();
    /**
     * N is for number of points of SLAU
     * @type int
     */
    // int N = 15;

    /*
    * Getting inputs A and B
    */
    vector<vector<double> > A((N - 2) * (N - 2), vector<double>((N - 2) * (N - 2), 0));
    readMatr(A);
    vector<vector<double> > B(N, vector<double>(N, 0));
    vector<double> Tau(1, 0);
    vector<vector<double> > firstAppr(N, vector<double>(N, 0));
    vector<vector<double> > tempAppr(N, vector<double>(N, 0));
    firstApprSet(firstAppr);
    readVector(B);
    alglib::real_2d_array matrix;
    matrix.setcontent((N - 2) * (N - 2), (N - 2) * (N - 2), arrToRealArr(A));
    double eps = 0.00001;
    /*
    *creating another parts
    *wr - целые части собственных чисел
    *wi - мнимые части собственных чисел
    *vl - собственный левый вектор
    *vr - собственный правый вектор
    */
    alglib::real_1d_array wr;
    alglib::real_1d_array wi;
    alglib::real_2d_array vl;
    alglib::real_2d_array vr;
    /*
    * расчет собственных чисел
    */
    alglib::rmatrixevd(matrix, (N - 2) * (N - 2), 0, wr, wi, vl, vr);
    double AlphaMax = findMaxRealArr(wr);
    double AlphaMin = findMinRealArr(wr);
    Tau[0] = 2. / (AlphaMax + AlphaMin);
    double ksi = AlphaMin / AlphaMax;
    double ro0 = (1. - ksi) / (1. + ksi);
    double ro1 = (1. - sqrt(ksi)) / (1. + sqrt(ksi));
    int maxIter = findMaxIter(eps, ksi);
    maxIter = maxIter * 2;
    vector<double> optTau(1, 1);
    vector<double> duo(0);
    decToDuo(duo, maxIter);
    calculateOptTau(optTau, duo);
    for (int i = 1; i < maxIter + 1; ++i) Tau.push_back(nextTau(Tau, ro0, maxIter, optTau));


    /*
    *main loop here
    */
    double *temp = new double [N * N - 4 * N + 4];
    double *all = new double [N * N];
    double *b = new double [N * N - 4 * N + 4];
    double *fa = new double [N * N - 4 * N + 4];
    double *taum = new double [maxIter + 1];
    double *d_a, *d_b, *d_c, *d_d, *d_g;
    int size = sizeof(double);

    hipMalloc((void **)&d_a, size * (N * N - 4 * N + 4));
    hipMalloc((void **)&d_b, size * (N * N - 4 * N + 4));
    hipMalloc((void **)&d_g, size * (N * N));
    hipMalloc((void **)&d_c, size * (maxIter + 1));
    hipMalloc((void **)&d_d, size * (N * N - 4 * N + 4));

    for (int j = 1; j < N - 1; j++) {
        for (int k = 1; k < N - 1; k++) {
            temp[(j - 1) * (N - 2) + (k - 1)] = 0;
            b[(j - 1) * (N - 2) + (k - 1)] = B[j][k];
            fa[(j - 1) * (N - 2) + (k - 1)] = firstAppr[j][k];
        }
    }
    for (int j = 0; j < N ; j++) {
        for (int k = 0; k < N; k++) {
            all[j * N + k] = firstAppr[j][k];
        }
    }
    for (int i = 0; i < maxIter + 1; i++) {
        taum[i] = Tau[i];
    }
    hipMemcpy(d_a, b, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
    hipMemcpy(d_d, fa, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
    hipMemcpy(d_c, taum, size * (maxIter + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_g, all, size * (N * N), hipMemcpyHostToDevice);
    hipMemcpy(d_b, temp, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
    double timeChecker = dsecnd();
    // char aster;
    for (int i = 1; i < maxIter + 1; ++i) {
        // cout<<"The "<<i<<" iter"<<endl;
        myshab<<<N - 2, N - 2>>>(d_b, N * N - 4 * N + 4, d_g);
        // hipMemcpy(temp, d_b, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
        // cout<<endl<<"The temp from GPU is"<<endl;
        // outVector(temp, N * N - 4 * N + 4);
        // Shablon(firstAppr, temp);
        // cout<<"The temp is"<<endl;
        // outVector(temp, N * N - 4 * N + 4);
        // cin>>aster;
        // hipMemcpy(d_b, temp, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
        mykernel<<<N - 2, N - 2>>>(d_a, d_b, d_c, d_d, N * N - 4 * N + 4, i, d_g);
        // hipMemcpy(fa, d_d, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
        // hipMemcpy(all, d_g, size * (N * N), hipMemcpyDeviceToHost);
        // for (int j = 1; j < N - 1; j++) {
        //     for (int k = 1; k < N - 1; k++) {
        //         firstAppr[j][k] = fa[(j - 1) * (N - 2) + (k - 1)];
        //     }
        // }
        // cout<<endl<<"fa"<<endl;
        // outMatr(firstAppr);
        // cout<<"ALLL"<<endl;
        // outVector(all, N * N);
        // cout<<endl;
    }
    double tMain = dsecnd() - timeChecker;
    hipMemcpy(fa, d_d, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
    for (int j = 1; j < N - 1; j++) {
        for (int k = 1; k < N - 1; k++) {
            firstAppr[j][k] = fa[(j - 1) * (N - 2) + (k - 1)];
        }
    }

    // for (int i = 0; i < firstAppr.size(); i++) {
    //     firstAppr[i] /= ((firstAppr.size() - 1) * (firstAppr.size() - 1));
    // }
    hipFree(d_a);
    hipFree(d_b);
    // hipFree(d_c);
    hipFree(d_d);
    /*
    * outing
    */
    firstApprSet(tempAppr);
    cout<< "The N is : " << N << endl;
    cout<<"The A(shorted) Is:"<<endl;
    outMatr(A);
    cout<<"The B(shorted) Is:"<<endl;
    outMatr(B);
    cout<<"The duo(shorted) Is:"<<endl;
    outVector(duo);
    cout<<"The opt(shorted) Is:"<<endl;
    outVector(optTau);
    cout<<"The first appr Is:"<<endl;
    outMatr(tempAppr);
    cout<<"The last approximation Is:"<<endl;
    outMatr(firstAppr);
    cout<<"The Max alpha Is:"<<endl;
    cout<<AlphaMax<<endl;
    cout<<"The Min alpha Is:"<<endl;
    cout<<AlphaMin<<endl;
    cout<<"The Tau is:"<<endl;
    outVector(Tau);
    cout<<"The ksi is:"<<endl;
    cout<<ksi<<endl;
    cout<<"The ro0 is:"<<endl;
    cout<<ro0<<endl;
    cout<<"The ro1 is:"<<endl;
    cout<<ro1<<endl;
    cout<<"The maxIter is:"<<endl;
    cout<<maxIter<<endl;
    cout<<"The time is:"<<endl;
    cout<< dsecnd() - t0 <<" s"<<endl;
    cout<<"The time of main is:"<<endl;
    cout<< tMain <<" s"<<endl;
    cout<<"The 1 1 is:"<<endl;
    cout<< firstAppr[1][1]<<endl;
    cout<<"The 2 2 is:"<<endl;
    cout<< firstAppr[2][2]<<endl;
    cout<<"The N - 2 N - 2 is:"<<endl;
    cout<< firstAppr[firstAppr.size() - 2][firstAppr.size() - 2]<<endl;
    cout<<"The N - 3 N - 3 is:"<<endl;
    cout<< firstAppr[firstAppr.size() - 3][firstAppr.size() - 3]<<endl;
    return 0;
}
