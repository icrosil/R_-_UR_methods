#include "hip/hip_runtime.h"
//compile g++ -o filename file.cpp and run filename
#include <iostream>
#include <math.h>
#include "vector"
#include "../lib/alglib/src/ap.h"
#include "../lib/alglib/src/alglibmisc.h"
#include "../lib/alglib/src/alglibinternal.h"
#include "../lib/alglib/src/linalg.h"
#include "../lib/alglib/src/statistics.h"
#include "../lib/alglib/src/dataanalysis.h"
#include "../lib/alglib/src/specialfunctions.h"
#include "../lib/alglib/src/solvers.h"
#include "../lib/alglib/src/optimization.h"
#include "../lib/alglib/src/diffequations.h"
#include "../lib/alglib/src/fasttransforms.h"
#include "../lib/alglib/src/integration.h"
#include "../lib/alglib/src/interpolation.h"
#include "../lib/alglib/src/ap.cpp"
#include "../lib/alglib/src/linalg.cpp"
#include "../lib/alglib/src/alglibmisc.cpp"
#include "../lib/alglib/src/alglibinternal.cpp"
#include "../lib/alglib/src/statistics.cpp"
#include "../lib/alglib/src/dataanalysis.cpp"
#include "../lib/alglib/src/specialfunctions.cpp"
#include "../lib/alglib/src/solvers.cpp"
#include "../lib/alglib/src/optimization.cpp"
#include "../lib/alglib/src/diffequations.cpp"
#include "../lib/alglib/src/fasttransforms.cpp"
#include "../lib/alglib/src/integration.cpp"
#include "../lib/alglib/src/interpolation.cpp"
#include <string>
#include <ctime>
#include <mkl.h>

using namespace std;
using namespace alglib;
using namespace alglib_impl;

void outVector (vector<double> B){
    // int additions = B.size() / 4;
    int additions = 1;
    cout<< B[0] << " ";
    for (int i = additions; i < B.size() - 1; i += additions){
        cout<< B[i] << " ";
    }
    cout<< B[B.size() - 1] << " ";
    cout<<endl;
}
void outVector (double* B, int N) {
    // cout<<"in outVector"<<endl;
    // int additions = B.size() / 4;
    int additions = 1;
    cout<< B[0] << " ";
    for (int i = additions; i < N - 1; i += additions){
        cout<< B[i] << " ";
    }
    cout<< B[N - 1] << " ";
    cout<<endl;
    cout<<"out outVector"<<endl;
}
void outMatr (vector<vector<double> > A){
    // int additions = A.size() / 4;
    int additions = 1;
    outVector(A[0]);
    for (int i = additions; i < A.size() - 1; i += additions){
        outVector(A[i]);
    }
    outVector(A[A.size() - 1]);
}
double F (double x, double y, double N) {
    // cout<<x<<" "<<y<<endl;
    return (2 * sin(y) - x * x * sin(y)) / N;
}
double U (double x, double y) {
    // cout<<x<<" "<<y<<endl;
    return x * x * sin(y) + 1;
}
void readMatr (vector<vector<double> > &A){
    A[0][0] = -4. ;
    A[0][1] = 1. ;
    A[A.size() - 1][A.size() - 1] = -4.;
    A[A.size() - 1][A.size() - 2] = 1.;
    for (int i = 1; i < A.size() - 1; i++) {
        A[i][i - 1] = 1.;
        A[i][i] = -4.;
        A[i][i + 1] = 1.;
    }
}
void readVector (vector<vector<double> >& B){
    for (int i = 0; i < B.size(); i++) {
        B[i][0] = U(i / (double) (B.size() - 1), 0);
        B[0][i] = U(0, i / (double) (B.size() - 1));
        B[B.size() - 1][i] = U(1, i / (double) (B.size() - 1));
        B[i][B.size() - 1] = U(i / (double) (B.size() - 1), 1);
    }
    for (int i = 1; i < B.size() - 1; ++i) {
        for (int j = 1; j < B.size() - 1; ++j) {
            B[i][j] = F(i / (double) (B.size() - 1), j / (double) (B.size() - 1), (B.size() - 1) * (B.size() - 1));
        }
    }
}
double* arrToRealArr (vector<vector<double> >const &A){
    double * local;
    local = new double[A.size() * A.size()];
    for (int i = 0; i < A.size(); ++i){
        for (int j = 0; j < A[i].size();++j){
            local[i * A.size() + j] = A[i][j];
        }
    }
    return local;
}
void outReal1Array (alglib::real_1d_array wr) {
    for (int i = 0; i < wr.length(); ++i )
    cout<<wr[i]<<" ";
    cout<<endl;
}
double findMaxRealArr (alglib::real_1d_array const wr) {
    double max = fabs(wr[0]);
    for (int i = 1; i < wr.length(); ++i) {
        if (fabs(wr[i]) > max) max = fabs(wr[i]);
    }
    return max;
}
double findMinRealArr (alglib::real_1d_array const wr) {
    double min = fabs(wr[0]);
    for (int i = 1; i < wr.length(); ++i) {
        if (fabs(wr[i]) < min) min = fabs(wr[i]);
    }
    return min;
}

double nextTau (vector<double> Tau, double ro0, int n, vector<double> optTau) {
    double tk = cos (( optTau[Tau.size() - 1]) * M_PI / (2 * n));
    // return Tau[0] / (1 + ro0 * tk);
    // if ((Tau[0] / (1 + ro0 * tk) - Tau[Tau.size() - 1]) > 1.) {
        // counter--;
        // return Tau[counter];
    // } else {
        // counter = Tau.size();
        return Tau[0] / (1 + ro0 * tk);
    // }
    // return 2 / (max + min + (max - min) * tk);
}
int findMaxIter (double eps, double ksi) {
    // return ceil(log (2. / eps) / (2. * sqrt(ksi)));
    return ceil(log (2. / eps) / (2. * sqrt(ksi)));
}
void firstApprSet(vector<vector<double> >& B) {
    for (int i = 0; i < B.size(); i++) {
        B[i][0] = U(i / (double) (B.size() - 1), 0);
        B[0][i] = U(0, i / (double) (B.size() - 1));
        B[B.size() - 1][i] = U(1, i / (double) (B.size() - 1));
        B[i][B.size() - 1] = U(i / (double) (B.size() - 1), 1);
    }
    for (int i = 1; i < B.size() - 1; ++i) {
        for (int j = 1; j < B.size() - 1; ++j) {
            B[i][j] = F(i / (double) (B.size() - 1), j / (double) (B.size() - 1), (B.size() - 1) * (B.size() - 1)) / 2.;
        }
    }
}
double aMulX(vector<vector<double> > A, vector<double> X, int j){
    double res = 0;
    for (int i = 0; i < A.size(); ++i){
        res += A[j][i] * X[i];
    }
    return res;
}
void decToDuo(vector<double> &duo, int maxIter) {
    // while (maxIter > 0) {
    //     duo.push_back(maxIter % 2);
    //     maxIter /= 2;
    // }
    vector<double> temp(0);
    while (maxIter > 1) {
        if (maxIter % 2 != 0) {
                temp.push_back(1);
                maxIter--;
        } else {
            temp.push_back(0);
            maxIter /= 2;
        }
    }
    for (int j = 0; j < temp.size(); j++) {
        /* code */
        duo.push_back(temp[temp.size() - j - 1]);
    }
}
void calculateOptTau(vector<double> &optTau, vector<double> duo) {
    vector<double> temp(0);
    double coord = 0;
    double number = 1;
    // duo[0] = 0;
    // duo.erase(duo.begin() + duo.size() - 1);
    for (int i = 0; i < duo.size(); i++) {
        temp.clear();
        coord = 0;
        if (duo[i] == 0) {
            number *= 2;
            if ((i + 1 == duo.size()) || (duo[i + 1] == 0)) {
                /*normal*/
                coord = 0;
            } else {
                /*4m+2*/
                coord = 2;
            }
            for (int j = 0; j < optTau.size(); j++) {
                temp.push_back(optTau[j]);
                temp.push_back(4 * optTau.size() + coord - optTau[j]);
            }
        } else {
            number += 1;
            /*+1*/
            for (int j = 0; j < optTau.size(); j++) {
                temp.push_back(optTau[j]);
            }
            temp.push_back(number);
        }
        optTau.clear();
        for (int j = 0; j < temp.size(); j++) {
            optTau.push_back(temp[j]);
        }
        outVector(optTau);
    }
}
double* aMulXVector(vector<vector<double> > A, vector<double> X){
    double *res = new double [X.size()];
    for (int j = 0; j < X.size(); j++) {
        res[j] = 0;
    }
    for (int j = 0; j < A.size(); ++j){
        for (int i = 0; i < A.size(); ++i){
            res[j] += A[j][i] * X[i];
        }
    }
    return res;
}
void Shablon(vector<vector<double> > X, double * &res) {
    // cout<<"size is "<<size;
    // double *res = new double [size];
    // cout<<"in Shabo"<<endl;
    // for (int j = 0; j < size; j++) {
    //     res[j] = 0;
    // }
    cout<<"in Shabo"<<endl;
    for (int j = 1; j < X.size() - 1; ++j) {
        for (int i = 1; i < X.size() - 1; ++i) {
            res[(i - 1) * (X.size() - 2) + (j - 1) ] = X[i + 1][j] + X[i - 1][j] + X[i][j + 1] + X[i][j - 1] - 4. * X[i][j];
        }
    }
    cout<<"out Shabo"<<endl;
    // return res;
}

/**
 * CUDA functions
 */

__global__ void mykernel (double *a, double *b, double c, double *d, int n) {
//B, Shablon, Tau[i - 1], firstAppr
//tempAppr[j] = (B[j] - aMulX(A, firstAppr, j)) * Tau[i - 1] + firstAppr[j];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        d[index] = (-a[index] + b[index]) * (c) + d[index];
    }
}



int main() {
    /**
     * t0 is for documenting whole processing time
     * @type double
     */
    double t0 = dsecnd();
    /**
     * N is for number of points of SLAU
     * @type int
     */
    int N = 30;

    /*
    * Getting inputs A and B
    */
    vector<vector<double> > A(N, vector<double>(N, 0));
    readMatr(A);
    vector<vector<double> > B(N, vector<double>(N, 0));
    vector<double> Tau(1, 0);
    vector<vector<double> > firstAppr(N, vector<double>(N, 0));
    vector<vector<double> > tempAppr(N, vector<double>(N, 0));
    firstApprSet(firstAppr);
    readVector(B);
    alglib::real_2d_array matrix;
    matrix.setcontent(N, N, arrToRealArr(A));
    double eps = 0.00001;
    /*
    *creating another parts
    *wr - целые части собственных чисел
    *wi - мнимые части собственных чисел
    *vl - собственный левый вектор
    *vr - собственный правый вектор
    */
    alglib::real_1d_array wr;
    alglib::real_1d_array wi;
    alglib::real_2d_array vl;
    alglib::real_2d_array vr;
    /*
    * расчет собственных чисел
    */
    alglib::rmatrixevd(matrix, N, 0, wr, wi, vl, vr);
    double AlphaMax = findMaxRealArr(wr);
    double AlphaMin = findMinRealArr(wr);
    Tau[0] = 2. / (AlphaMax + AlphaMin);
    double ksi = AlphaMin / AlphaMax;
    double ro0 = (1. - ksi) / (1. + ksi);
    double ro1 = (1. - sqrt(ksi)) / (1. + sqrt(ksi));
    int maxIter = findMaxIter(eps, ksi);
    maxIter = maxIter * N * (int) (N / 5);
    vector<double> optTau(1, 1);
    vector<double> duo(0);
    decToDuo(duo, maxIter);
    calculateOptTau(optTau, duo);
    for (int i = 1; i < maxIter + 1; ++i) Tau.push_back(nextTau(Tau, ro0, maxIter, optTau));


    /*
    *main loop here
    */
    double *temp = new double [N * N - 4 * N + 4];
    double *b = new double [N * N - 4 * N + 4];
    double *fa = new double [N * N - 4 * N + 4];
    double *d_a, *d_b, *d_d;
    int size = sizeof(double);

    hipMalloc((void **)&d_a, size * (N * N - 4 * N + 4));
    hipMalloc((void **)&d_b, size * (N * N - 4 * N + 4));
    // hipMalloc((void **)&d_c, size);
    hipMalloc((void **)&d_d, size * (N * N - 4 * N + 4));

    for (int j = 1; j < N - 1; j++) {
        for (int k = 1; k < N - 1; k++) {
            temp[(j - 1) * (N - 2) + (k - 1)] = 0;
            b[(j - 1) * (N - 2) + (k - 1)] = B[j][k];
            fa[(j - 1) * (N - 2) + (k - 1)] = firstAppr[j][k];
        }
    }
    hipMemcpy(d_a, b, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
    hipMemcpy(d_d, fa, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
    double timeChecker = dsecnd();;
    for (int i = 1; i < maxIter + 1; ++i) {
        cout<<"The "<<i<<" iter"<<endl;
        Shablon(firstAppr, temp);
        // cout<<"The temp is"<<endl;
        // outVector(temp, N * N - 4 * N + 4);
        // cout<<endl<<Tau[i]<<" "<<&Tau[i];
        // hipMemcpy(d_c, &Tau[i], size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, temp, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
        mykernel<<<(N * N - 4 * N + 4) / (N - 2), N - 2>>>(d_a, d_b, Tau[i], d_d, N * N - 4 * N + 4);
        hipMemcpy(fa, d_d, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
        for (int j = 1; j < N - 1; j++) {
            for (int k = 1; k < N - 1; k++) {
                firstAppr[j][k] = fa[(j - 1) * (N - 2) + (k - 1)];
            }
        }
        outMatr(firstAppr);
        cout<<endl;
    }
    cout<<dsecnd() - timeChecker<<" time for loop"<<endl;

    // for (int i = 0; i < firstAppr.size(); i++) {
    //     firstAppr[i] /= ((firstAppr.size() - 1) * (firstAppr.size() - 1));
    // }
    hipFree(d_a);
    hipFree(d_b);
    // hipFree(d_c);
    hipFree(d_d);
    /*
    * outing
    */
    firstApprSet(tempAppr);
    cout<< "The N is : " << N << endl;
    cout<<"The A(shorted) Is:"<<endl;
    outMatr(A);
    cout<<"The B(shorted) Is:"<<endl;
    outMatr(B);
    cout<<"The duo(shorted) Is:"<<endl;
    outVector(duo);
    cout<<"The opt(shorted) Is:"<<endl;
    outVector(optTau);
    cout<<"The first appr Is:"<<endl;
    outMatr(tempAppr);
    cout<<"The last approximation Is:"<<endl;
    outMatr(firstAppr);
    cout<<"The Max alpha Is:"<<endl;
    cout<<AlphaMax<<endl;
    cout<<"The Min alpha Is:"<<endl;
    cout<<AlphaMin<<endl;
    cout<<"The Tau is:"<<endl;
    outVector(Tau);
    cout<<"The ksi is:"<<endl;
    cout<<ksi<<endl;
    cout<<"The ro0 is:"<<endl;
    cout<<ro0<<endl;
    cout<<"The ro1 is:"<<endl;
    cout<<ro1<<endl;
    cout<<"The maxIter is:"<<endl;
    cout<<maxIter<<endl;
    cout<<"The time is:"<<endl;
    cout<< dsecnd() - t0 <<" s"<<endl;
    cout<<"The 1 1 is:"<<endl;
    cout<< firstAppr[1][1]<<endl;
    cout<<"The 2 2 is:"<<endl;
    cout<< firstAppr[2][2]<<endl;
    cout<<"The N - 2 N - 2 is:"<<endl;
    cout<< firstAppr[firstAppr.size() - 2][firstAppr.size() - 2]<<endl;
    cout<<"The N - 3 N - 3 is:"<<endl;
    cout<< firstAppr[firstAppr.size() - 3][firstAppr.size() - 3]<<endl;
    return 0;
}
