#include "hip/hip_runtime.h"
// Copyright 2015-2017 Illia Olenchenko

#include <math.h>
#include <iostream>
#include "vector"
#include "../lib/alglib/src/ap.h"
#include "../lib/alglib/src/alglibmisc.h"
#include "../lib/alglib/src/alglibinternal.h"
#include "../lib/alglib/src/linalg.h"
#include "../lib/alglib/src/statistics.h"
#include "../lib/alglib/src/dataanalysis.h"
#include "../lib/alglib/src/specialfunctions.h"
#include "../lib/alglib/src/solvers.h"
#include "../lib/alglib/src/optimization.h"
#include "../lib/alglib/src/diffequations.h"
#include "../lib/alglib/src/fasttransforms.h"
#include "../lib/alglib/src/integration.h"
#include "../lib/alglib/src/interpolation.h"
#include "../utils/out.h"
#include "../utils/functions.h"
#include "../utils/init.h"
#include "../utils/transform.h"
#include "../utils/richardson.h"
#include "../utils/tools.h"
#include <string>
#include <ctime>
#include <mkl.h>
#include <stdio.h>

using namespace std;
using namespace alglib;
using namespace alglib_impl;

/**
 * CUDA functions
 */

/**
 * N is for number of points of SLAU
 * should be N % 2 == 0 for correct split
 * @type int
 */
#ifndef N
#define N 4
#endif
#ifndef GPU
#define GPU 2
#endif

__device__ int barrier = N - 2;
__device__ int blocks = N - 2;

__global__ void myshab(double *temp, int n, double *all) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  // #if __CUDA_ARCH__ >= 200
  //   printf("%d col \n", col);
  //   printf("%d row \n", row);
  // #endif
  int index = row * n + col;
  if (index >= n * n) return;
  int lindex = index + N + 1 + 2 * (int) (index / (N - 2));
  temp[index] = -4 * all[lindex] + all[lindex - N] + all[lindex + N] + all[lindex - 1] + all[lindex + 1];
}

// B, Shablon, Tau, firstAppr, iteration number
__global__ void mykernel(double *a, double *b, double *c, double *d, int n, int i, double *all) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int index = row * n + col;
    if (index >= n * n) return;
    int lindex = index + N + 1 + 2 * (int) (index / (N - 2));
    d[index] = (-a[index] + b[index]) * c[i] + d[index];
    all[lindex] = d[index];
}


int main() {
  /**
   * t0 is for documenting whole processing time
   * @type double
   */
  double t0 = dsecnd();
  // multi GPUs
  vector<int> cudas(GPU);
  for (size_t i = 0; i < GPU; i++) {
    cudas[i] = i;
  }
  // inner Ns
  int n_inner = (N - 2) * (N - 2);
  int n_splitted_inner = n_inner / GPU;

  /*
  * Getting inputs A and B
  */
  vector<vector<double> > A(n_inner, vector<double>(n_inner, 0));
  readMatr(A);
  vector<vector<double> > B(N, vector<double>(N, 0));
  vector<double> Tau(1, 0);
  vector<vector<double> > firstAppr(N, vector<double>(N, 0));
  vector<vector<double> > tempAppr(N, vector<double>(N, 0));
  firstApprSet(firstAppr);
  readVector(B);
  alglib::real_2d_array matrix;
  matrix.setcontent(n_inner, n_inner, arrToRealArr(A));
  double eps = 0.01;
  /*
  *creating another parts
  *wr - целые части собственных чисел
  *wi - мнимые части собственных чисел
  *vl - собственный левый вектор
  *vr - собственный правый вектор
  */
  alglib::real_1d_array wr;
  alglib::real_1d_array wi;
  alglib::real_2d_array vl;
  alglib::real_2d_array vr;
  /*
  * расчет собственных чисел
  */
  alglib::rmatrixevd(matrix, n_inner, 0, wr, wi, vl, vr);
  double AlphaMax = findMaxRealArr(wr);
  double AlphaMin = findMinRealArr(wr);
  Tau[0] = 2. / (AlphaMax + AlphaMin);
  double ksi = AlphaMin / AlphaMax;
  std::cout << ksi << "ksi" << '\n';  // is it important to calculate n*n alphas?
  double ro0 = (1. - ksi) / (1. + ksi);
  double ro1 = (1. - sqrt(ksi)) / (1. + sqrt(ksi));
  int maxIter = findMaxIter(eps, ksi);
  maxIter = maxIter * 2;
  vector<double> optTau(1, 1);
  vector<double> duo(0);
  decToDuo(duo, maxIter);
  calculateOptTau(optTau, duo);
  for (int i = 1; i < maxIter + 1; ++i) Tau.push_back(nextTau(Tau, ro0, maxIter, optTau));


  /*
  *main loop here
  */
  double **temp = new double*[(int)GPU];
  double *all = new double[N * N];
  double **b = new double*[(int)GPU];
  double **fa = new double*[(int)GPU];
  double *taum = new double[maxIter + 1];
  for (size_t i = 0; i < GPU; i++) {
    temp[i] = new double[(int)n_splitted_inner];
    b[i] = new double[(int)n_splitted_inner];
    fa[i] = new double[(int)n_splitted_inner];
  }

  double *d_a[GPU],
    *d_b[GPU],
    *d_c[GPU],
    *d_d[GPU],
    *d_g[GPU];
  int size = sizeof(double);

  for (size_t i = 0; i < GPU; i++) {
    hipSetDevice(cudas[i]);
    hipMalloc((void **)&d_a[i], size * (n_splitted_inner));
    hipMalloc((void **)&d_b[i], size * (n_splitted_inner));
    hipMalloc((void **)&d_d[i], size * (n_splitted_inner));
    hipMalloc((void **)&d_g[i], size * (N * N));
    hipMalloc((void **)&d_c[i], size * (maxIter + 1));
  }

  for (size_t i = 0; i < GPU; i++) {
    int plus = i * ((int)(N / 2) - 1);
    for (int j = 1; j < (int)(N / 2); j++) {
      for (int k = 1; k < N - 1; k++) {
        temp[i][(j - 1) * (int)((N - 2) / 2) + (k - 1)] = 0;
        b[i][(j - 1) * (int)((N - 2) / 2) + (k - 1)] = B[j + plus][k];
        fa[i][(j - 1) * (int)((N - 2) / 2) + (k - 1)] = firstAppr[j + plus][k];
      }
    }
  }
  for (int j = 0; j < N ; j++) {
    for (int k = 0; k < N; k++) {
      all[j * N + k] = firstAppr[j][k];
    }
  }
  for (int i = 0; i < maxIter + 1; i++) {
    taum[i] = Tau[i];
  }
  // hipMemcpy(d_a, b, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
  // hipMemcpy(d_d, fa, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
  // hipMemcpy(d_c, taum, size * (maxIter + 1), hipMemcpyHostToDevice);
  // hipMemcpy(d_g, all, size * (N * N), hipMemcpyHostToDevice);
  // hipMemcpy(d_b, temp, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
  // double timeChecker = dsecnd();
  // // char aster;
  // dim3 threadsPerBlock(16, 16);
  // dim3 numBlocks(max((N - 2) / threadsPerBlock.x, 1), max((N - 2) / threadsPerBlock.y, 1));
  // for (int i = 1; i < maxIter + 1; ++i) {
  //   myshab <<<numBlocks, threadsPerBlock>>>(d_b, N - 2, d_g);
  //   mykernel <<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, d_d, N - 2, i, d_g);
  //
  //   // cout <<"The " <<i <<" iter" <<endl;
  //   // hipMemcpy(temp, d_b, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
  //   // cout <<endl <<"The temp from GPU is" <<endl;
  //   // outVector(temp, N * N - 4 * N + 4);
  //   // Shablon(firstAppr, temp);
  //   // cout <<"The temp is" <<endl;
  //   // outVector(temp, N * N - 4 * N + 4);
  //   // cin >> aster;
  //   // hipMemcpy(d_b, temp, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
  //   // hipMemcpy(fa, d_d, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
  //   // hipMemcpy(all, d_g, size * (N * N), hipMemcpyDeviceToHost);
  //   // for (int j = 1; j < N - 1; j++) {
  //   //   for (int k = 1; k < N - 1; k++) {
  //   //     firstAppr[j][k] = fa[(j - 1) * (N - 2) + (k - 1)];
  //   //   }
  //   // }
  //   // cout <<endl <<"fa" <<endl;
  //   // outMatr(firstAppr);
  //   // cout <<"ALLL" <<endl;
  //   // outVector(all, N * N);
  //   // cout <<endl;
  // }
  // double tMain = dsecnd() - timeChecker;
  // hipMemcpy(fa, d_d, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
  // for (int j = 1; j < N - 1; j++) {
  //   for (int k = 1; k < N - 1; k++) {
  //     firstAppr[j][k] = fa[(j - 1) * (N - 2) + (k - 1)];
  //   }
  // }
  // hipFree(d_a);
  // hipFree(d_b);
  // // hipFree(d_c);
  // hipFree(d_d);
  /*
  * outing
  */
  // firstApprSet(tempAppr);
  // cout << "The N is : " << N << endl;
  // cout <<"The A(shorted) Is:" <<endl;
  // outMatr(A);
  // cout <<"The B(shorted) Is:" <<endl;
  // outMatr(B);
  // cout <<"The duo(shorted) Is:" <<endl;
  // outVector(duo);
  // cout <<"The opt(shorted) Is:" <<endl;
  // outVector(optTau);
  // cout <<"The first appr Is:" <<endl;
  // outMatr(tempAppr);
  // cout <<"The last approximation Is:" <<endl;
  // outMatr(firstAppr);
  // cout <<"The Max alpha Is:" <<endl;
  // cout <<AlphaMax <<endl;
  // cout <<"The Min alpha Is:" <<endl;
  // cout <<AlphaMin <<endl;
  // cout <<"The Tau is:" <<endl;
  // outVector(Tau);
  // cout <<"The ksi is:" <<endl;
  // cout <<ksi <<endl;
  // cout <<"The ro0 is:" <<endl;
  // cout <<ro0 <<endl;
  // cout <<"The ro1 is:" <<endl;
  // cout <<ro1 <<endl;
  // cout <<"The maxIter is:" <<endl;
  // cout <<maxIter <<endl;
  // cout <<"The time is:" <<endl;
  // cout << dsecnd() - t0 <<" s" <<endl;
  // cout <<"The time of main is:" <<endl;
  // cout << tMain <<" s" <<endl;
  // cout <<"The 1 1 is:" <<endl;
  // cout << firstAppr[1][1] <<endl;
  // cout <<"The 2 2 is:" <<endl;
  // cout << firstAppr[2][2] <<endl;
  // cout <<"The N - 2 N - 2 is:" <<endl;
  // cout << firstAppr[firstAppr.size() - 2][firstAppr.size() - 2] <<endl;
  // cout <<"The N - 3 N - 3 is:" <<endl;
  // cout << firstAppr[firstAppr.size() - 3][firstAppr.size() - 3] <<endl;
  return 0;
}
