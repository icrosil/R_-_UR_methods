#include "hip/hip_runtime.h"
// Copyright 2015-2017 Illia Olenchenko

#include <iostream>
#include <math.h>
#include "vector"
#include "../lib/alglib/src/ap.h"
#include "../lib/alglib/src/alglibmisc.h"
#include "../lib/alglib/src/alglibinternal.h"
#include "../lib/alglib/src/linalg.h"
#include "../lib/alglib/src/statistics.h"
#include "../lib/alglib/src/dataanalysis.h"
#include "../lib/alglib/src/specialfunctions.h"
#include "../lib/alglib/src/solvers.h"
#include "../lib/alglib/src/optimization.h"
#include "../lib/alglib/src/diffequations.h"
#include "../lib/alglib/src/fasttransforms.h"
#include "../lib/alglib/src/integration.h"
#include "../lib/alglib/src/interpolation.h"
#include "../utils/out.h"
#include "../utils/functions.h"
#include "../utils/init.h"
#include "../utils/transform.h"
#include "../utils/upper_relaxation.h"
#include "../utils/tools.h"
#include <string>
#include <ctime>
#include <mkl.h>

using namespace std;
using namespace alglib;
using namespace alglib_impl;

/**
 * CUDA functions
 */
#ifndef N
#define N 5
#endif

__device__ int blocks = N - 2;

__global__ void mykernel(double *rightSide, double wOpt, double *fa, double *diff, int n, double *all, int i, int j,
  int litN) {
  // tempAppr[j] = (B[j] - aMulX(A, firstAppr, j)) * Tau[i - 1] + firstAppr[j];

  int index = threadIdx.x;  //+ blockIdx.x * blockDim.x;
  int row = (i - j + 1) / 2;
  int index1 = row * litN + (((j - i) == 1)? j : litN) - 1 + index * (litN - 1);
  int lindex = index1 + N + 1 + 2 * (int) (index1 / (N - 2));
  if (index1 < n) {
    //  fa[index1] = index1;
    fa[index1] = (-rightSide[index1] + all[lindex - N] + all[lindex + N] + all[lindex - 1] + all[lindex + 1] -
      4 * (1 - 1. / wOpt) * all[lindex]) * wOpt / 4.;
    diff[index1] = fa[index1] - all[lindex];
    all[lindex] = fa[index1];
  }
}
__global__ void my_red_black_kernel(double *rightSide, double wOpt, double *fa, double *diff, int n, double *all,
  int litN, int first) {
  // tempAppr[j] = (B[j] - aMulX(A, firstAppr, j)) * Tau[i - 1] + firstAppr[j];

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = index + N + 1 + 2 * (int) (index / (N - 2));
  int row = (int)(index / litN);
  int str = index % litN;
  if (index < n && ((row + str) % 2 == first)) {
    // fa[index] = index;
    fa[index] = (-rightSide[index] + all[lindex - N] + all[lindex + N] + all[lindex - 1] + all[lindex + 1] -
      4 * (1 - 1. / wOpt) * all[lindex]) * wOpt / 4.;
    diff[index] = fa[index] - all[lindex];
    all[lindex] = fa[index];
  }
}

int main() {
  double t0 = dsecnd();
  /*
  *TODO: add CUDA improvements
  *эта часть задачи решает по матрице и правой части итерационный процесс верхних релаксаций
  */
  /*
  * N means matr size
  * A means main Matr
  * B means right vector
  */

  /*
  * Getting inputs A and B
  */
  vector<vector<double> > A((N - 2) * (N - 2), vector<double>((N - 2) * (N - 2), 0));
  readMatr(A);
  vector<vector<double> > B(N, vector<double>(N, 0));
  vector<vector<double> > firstAppr(N, vector<double>(N, 0));
  vector<vector<double> > changeAppr(N, vector<double>(N, 0));
  firstApprSet(firstAppr);
  readVector(B);
  double eps = 0.0001;
  double spectr;
  double wOpt;
  double maxDiff = 0;
  alglib::real_2d_array matrix;
  matrix.setcontent((N - 2) * (N - 2), (N - 2) * (N - 2), arrToRealArr(A));

  /*
  *creating another parts
  *wr - целые части собственных чисел
  *wi - мнимые части собственных чисел
  *vl - собственный левый вектор
  *vr - собственный правый вектор
  */
  alglib::real_1d_array wr;
  alglib::real_1d_array wi;
  alglib::real_2d_array vl;
  alglib::real_2d_array vr;
  /*
  * расчет собственных чисел
  */
  alglib::rmatrixevd(matrix, N, 0, wr, wi, vl, vr);

  /*
  *допустим что спектральынй радиус матрицы это максимальное собственное число (которые все норм должны быть) без модуля, так как все должны быть положительны
  */
  spectr = findMaxRealArr(wr);
  wOpt = wOptSet(A, spectr, 1. / N);

  /*
  *main loop here
  *если я правильно понял то новые вычисления нужно тут же использовать, исхожу из этого мнения
  */
  int k = 0;
  // char aber;
  double *rightSide = new double[N * N - 4 * N + 4];
  double *fa = new double[N * N - 4 * N + 4];
  double *all = new double[N * N];
  double *diff = new double[N * N - 4 * N + 4];
  double *d_rs, *d_fa, *d_all, *d_diff;
  int size = sizeof(double);

  hipMalloc((void **)&d_rs, size * (N * N - 4 * N + 4));
  hipMalloc((void **)&d_all, size * (N * N));
  hipMalloc((void **)&d_fa, size * (N * N - 4 * N + 4));
  hipMalloc((void **)&d_diff, size * (N * N - 4 * N + 4));
  for (int j = 1; j < N - 1; j++) {
    for (int k = 1; k < N - 1; k++) {
      rightSide[(j - 1) * (N - 2) + (k - 1)] = B[j][k];
      fa[(j - 1) * (N - 2) + (k - 1)] = firstAppr[j][k];
      diff[(j - 1) * (N - 2) + (k - 1)] = 1;
    }
  }
  for (int j = 0; j < N ; j++) {
    for (int k = 0; k < N; k++) {
      all[j * N + k] = firstAppr[j][k];
    }
  }
  hipMemcpy(d_rs, rightSide, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
  hipMemcpy(d_fa, fa, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
  hipMemcpy(d_all, all, size * (N * N), hipMemcpyHostToDevice);
  hipMemcpy(d_diff, diff, size * (N * N - 4 * N + 4), hipMemcpyHostToDevice);
  double timeChecker = dsecnd();
  int n = N - 2;
  for (int k = n; k > 0; --k) {
    for (int j = 0, i = 0; i < 2 * k - 1; i++) {
      if (i < n) {
        ++j;
      } else {
        --j;
      }
      mykernel<<<1, j>>>(d_rs, wOpt, d_fa, d_diff, N * N - 4 * N + 4, d_all, i, j, n);
    }
  }
  hipMemcpy(fa, d_fa, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
  outVector(fa, N * N - 4 * N + 4);
  // cin>>aber;
  hipMemcpy(diff, d_diff, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
  do {
    //   outVector(diff, N * N - 4 * N + 4);
    // diff[0] = 0;
    // hipMemcpy(d_diff, diff, size, hipMemcpyHostToDevice);
    cout << "The " << k << " iter" << endl;
    // copyVectors(firstAppr, changeAppr);
    // cout<<"change: "<<endl;
    // outMatr(changeAppr);
    // cout<<"fa: "<<endl;
    // outMatr(firstAppr);
    // cin>>aber;
    // for (int i = 0; i < A.size(); i++) {
    //     firstAppr[i] = firstAppr[i] + (B[i] - aMulX(A, firstAppr, i)) * wOpt / (DwL(A, i, wOpt));
    // }
    // for (int j = 1; j < N - 1; ++j) {
    //     for (int i = 1; i < N - 1; i++) {
    //               firstAppr[j][i] = (B[j][i] - (firstAppr[j][i + 1] + firstAppr[j][i - 1] +
    // firstAppr[j + 1][i] + firstAppr[j - 1][i] - 4 * firstAppr[j][i])) * wOpt / (DwL(A, i, wOpt)); + firstAppr[j][i];
    my_red_black_kernel<<<N - 2, N - 2>>>(d_rs, wOpt, d_fa, d_diff, N * N - 4 * N + 4, d_all, n, 1);
    // hipMemcpy(fa, d_fa, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
    // cout<<"Halfstep"<<endl;
    // outVector(fa, N * N - 4 * N + 4);
    my_red_black_kernel<<<N - 2, N - 2>>>(d_rs, wOpt, d_fa, d_diff, N * N - 4 * N + 4, d_all, n, 0);
    // for (int j = 0,i = 0; i < 2 * n - 1; i++) {
    //     if (i < n) {
    //         ++j;
    //     } else {
    //         --j;
    //     }
    //     mykernel<<<1, j>>>(d_rs, wOpt, d_fa, d_diff, N * N - 4 * N + 4, d_all, i, j, n);
    //   //   hipMemcpy(fa, d_fa, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
    //   //   cout<<"The fa is "<<endl;
    //   //   outVector(fa, N * N - 4 * N + 4);
    //   //   cin>>aber;
    // }
    // firstAppr[j][i] = (-B[j][i] + firstAppr[j + 1][i] + firstAppr[j - 1][i] + firstAppr[j][i - 1] +
    //   firstAppr[j][i + 1] - 4 * (1 - 1. / wOpt) * firstAppr[j][i]) * wOpt / 4.;
    //     }
    // }
    // for (int j = 1; j < N - 1; ++j) {
    //     for (int i = 1; i < N - 1; i++) {
    //         changeAppr[j][i] = fabs(firstAppr[j][i] - changeAppr[j][i]);
    //     }
    // }
    // hipMemcpy(fa, d_fa, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
    hipMemcpy(diff, d_diff, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
    // cout<<"The fa is "<<endl;
    // outVector(fa, N * N - 4 * N + 4);
    // cout<<endl;
    // outVector(diff, N * N - 4 * N + 4);
    maxDiff = findMaxInVector(diff, N * N - 4 * N + 4);
    // outVector(changeAppr);
    // cout<<findMaxInVector(changeAppr)<<endl;
    // maxDiff = findMaxInVector(changeAppr);
    // system("pause");
    ++k;
    cout << "Maxdiff is " << maxDiff << endl;
    // cin>>aber;
    if (maxDiff > 1000) {
      break;
    }
  } while (maxDiff > eps);
  timeChecker = dsecnd() - timeChecker;
  cout << "The iter is:" << endl;
  cout << k << endl;
  firstApprSet(changeAppr);
  hipMemcpy(fa, d_fa, size * (N * N - 4 * N + 4), hipMemcpyDeviceToHost);
  for (int j = 1; j < N - 1; j++) {
    for (int k = 1; k < N - 1; k++) {
      firstAppr[j][k] = fa[(j - 1) * (N - 2) + (k - 1)];
    }
  }
  hipFree(d_rs);
  hipFree(d_fa);
  // hipFree(d_c);
  hipFree(d_all);
  hipFree(d_diff);
  //   /*
  //   * outing
  //   */
  cout << "The Matr Is:" << endl;
  outMatr(A);
  cout << "The Vector Is:" << endl;
  outMatr(B);
  cout << "The first approximation Is:" << endl;
  outMatr(changeAppr);
  cout << "The epsilon Is:" << endl;
  cout << eps << endl;
  cout << "The Vector of ownValues:" << endl;
  outReal1Array(wr);
  cout << "The Spectr Is:" << endl;
  cout << spectr << endl;
  cout << "The wOpt Is:" << endl;
  cout << wOpt << endl;
  cout << "The result Is:" << endl;
  outMatr(firstAppr);
  cout << "The time is:" << endl;
  cout <<  dsecnd() - t0  << " s" << endl;
  cout << "The time of main is:" << endl;
  cout <<  timeChecker  << " s" << endl;
  cout << "The 1 1 is:" << endl;
  cout <<  firstAppr[1][1] << endl;
  cout << "The 2 2 is:" << endl;
  cout <<  firstAppr[2][2] << endl;
  cout << "The N - 2 N - 2 is:" << endl;
  cout <<  firstAppr[firstAppr.size() - 2][firstAppr.size() - 2] << endl;
  cout << "The N - 3 N - 3 is:" << endl;
  cout <<  firstAppr[firstAppr.size() - 3][firstAppr.size() - 3] << endl;
  return 0;
}
